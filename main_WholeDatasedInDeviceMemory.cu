#include <cstdlib>
#include <iostream>
#include <cstdio>
#include <stdio.h>
#include <stdlib.h>
#include <type_traits>
#include <memory>
#include <array>
#include <chrono>
#include <cfloat>
#include <time.h>
#include "KCPProblem.cuh"
#include "createSplitPoints.cuh"
#include "deviceFunctions.cuh"
#include "hostFunctions.cuh"
#include "kernelPhase2.cuh"
#include "kernelSample.cuh"
#include "BruteForceAlgorithm.cuh"
using namespace std::chrono; 

#define THREADS_PER_BLOCK 256

int main(int argc, char* argv[]){

    //we need the number of closest pairs (k) to be specified from command line
    if ( argc < 6 ) {
        
        std::cout << "Argument error. Please enter:\n";
        std::cout << "Argument 1: The number of the closest pairs:\n";
        std::cout << "Argument 2: The file of the first dataset P:\n";
        std::cout << "Argument 3: The file of the second dataset Q:\n";
        std::cout << "Argument 4: The number of the total partitions\n";
        std::cout << "Argument 5: The number of the total points of dataset P\n";
        std::cout << "Argument 6: The number of the total points of dataset Q\n";
        
        
        return 1;
    }

    try{

        int NofCP = atoi(argv[1]);
        int NofPartitions = atoi(argv[4]);
        int NofPointsP = atoi(argv[5]);
        int NofPointsQ = atoi(argv[6]);
        //NofSamplePartitions is actually the sample from which we will calculate the bound in phase1(kernelSample)
        int NofSamplePartitions = (int) NofPartitions*0.1;
        //SampleSize is for creating the split points(partitions)
        int SampleSizeP = (int) NofPointsP*0.3;
        int SampleSizeQ = (int) NofPointsQ*0.3;
        point_vector_t splitsP, splitsQ;
        point_vector_t datasetP, datasetQ;
        pair_vector_t neccArray ;
        point* h_datasetP = (point*)malloc(NofPointsP*sizeof(point));
        point* h_datasetQ = (point*)malloc(NofPointsQ*sizeof(point));
        //the maximum size of the necessary pairs of partitions to check 
        pair* h_neccArray = (pair*)malloc(NofPartitions*NofPartitions*sizeof(pair));
        ovpairs* neccPairs = (ovpairs*)malloc(NofSamplePartitions*sizeof(ovpairs));

        int *indexesOfSplitsP = new int[NofPartitions];
        int *indexesOfSplitsQ = new int[NofPartitions];
        int *indexesOfSplitsPOfSample = new int[NofSamplePartitions];
        int *indexesOfSplitsQOfSample = new int[NofSamplePartitions];
        
        // Default values
        std::string fileDatasetP("");
        std::string fileDatasetQ("");

        fileDatasetP = argv[2];
        fileDatasetQ = argv[3];

        std::unique_ptr<KCPProblem> query;
        
        //allocate the problem object depending
        //read the files and store the datasets 
        query.reset(new KCPProblem(fileDatasetP, fileDatasetQ, NofCP));
        
        std::unique_ptr<createSplitPoints> PointsP;
        std::unique_ptr<createSplitPoints> PointsQ;

        auto startTotal = high_resolution_clock::now();

        //find the splits points for each dataset P and Q
        PointsP.reset(new createSplitPoints(query->GetDatasetP(), NofPartitions, SampleSizeP));
        PointsQ.reset(new createSplitPoints(query->GetDatasetQ(), NofPartitions, SampleSizeQ));
        
        splitsP = PointsP->GetSplits();
        splitsQ = PointsQ->GetSplits();
        
        // sort the two datasets
        datasetP = query->GetDatasetP();
        PointsP->quicksort(datasetP, 0, NofPointsP-1);
        
        datasetQ = query->GetDatasetQ();
        PointsQ->quicksort(datasetQ, 0, NofPointsQ-1);
        
        //map the indexes of split points of partitions to each thread
        PointsP->mapPartitionsToThreads(datasetP, indexesOfSplitsP, splitsP, NofPartitions, NofPointsP);
        PointsQ->mapPartitionsToThreads(datasetQ, indexesOfSplitsQ, splitsQ, NofPartitions, NofPointsQ);
        
        // Invoke kernel for sample
        int threadsPerBlock = NofSamplePartitions;
        int blocksPerGrid = 1;
        int sizeofheap = threadsPerBlock*blocksPerGrid; 

        double* globalMaxKHeap = (double*)malloc(sizeofheap*NofCP*sizeof(double));
        

        if(NofPointsP == NofPointsQ){
            for(int i=0; i < NofPointsP; i++){
                h_datasetP[i] = datasetP[i];            
                h_datasetQ[i] = datasetQ[i];
            }
        }
        else{
            for(int i=0; i < NofPointsP; i++){
                h_datasetP[i] = datasetP[i];            
            }
            for(int i=0; i < NofPointsQ; i++){
                h_datasetQ[i] = datasetQ[i];
            }
        }

        for(int j=0; j < NofSamplePartitions; j++){
            indexesOfSplitsPOfSample[j] = indexesOfSplitsP[j];
            indexesOfSplitsQOfSample[j] = indexesOfSplitsQ[j];
            neccPairs[j].priority = -1;
        }

        // Allocate vectors in device memory
        double* d_globalMaxKHeap;
        point* d_datasetP;
        point* d_datasetQ;
        pair* d_neccArray;
        closestpairs* phase2_d_globalMaxKHeap;
        ovpairs* d_neccPairs;
        
        
        //start time of bound calculation
        auto start = high_resolution_clock::now(); 

        findOverlapPartitions(datasetP, datasetQ, indexesOfSplitsP, indexesOfSplitsQ, neccPairs, NofSamplePartitions, NofPartitions);

        hipMalloc(&d_globalMaxKHeap, sizeofheap*NofCP*sizeof(double));
        hipMalloc(&d_datasetP, NofPointsP*sizeof(point));
        hipMalloc(&d_datasetQ, NofPointsQ*sizeof(point));
        hipMalloc(&d_neccPairs, NofSamplePartitions*sizeof(ovpairs));
        
        // Copy vectors from host memory to device memory
        hipMemcpy(d_datasetP, h_datasetP, NofPointsP*sizeof(point), hipMemcpyHostToDevice);
        hipMemcpy(d_datasetQ, h_datasetQ, NofPointsQ*sizeof(point), hipMemcpyHostToDevice);
        hipMemcpy(d_neccPairs, neccPairs, NofSamplePartitions*sizeof(ovpairs), hipMemcpyHostToDevice);

        // Invoke kernel
        //start time of sample phase
        auto startPhaseSample = high_resolution_clock::now();
        kernelSample<<<blocksPerGrid, threadsPerBlock>>>(d_datasetP, d_datasetQ, NofCP, d_globalMaxKHeap, d_neccPairs);
        hipDeviceSynchronize();
        auto stopPhaseSample = high_resolution_clock::now();
        
        std::cout << "CLEAN Time difference(Sample phase) = " << std::chrono::duration_cast<std::chrono::milliseconds>(stopPhaseSample - startPhaseSample).count() << "[ms]" << std::endl;
        
        // Copy result from device memory to host memory
        hipMemcpy(globalMaxKHeap, d_globalMaxKHeap, sizeofheap*NofCP*sizeof(double), hipMemcpyDeviceToHost);

        auto stop = high_resolution_clock::now();
        //end time of bound calculation

        std::cout << "Time difference(Sample for bound) = " << std::chrono::duration_cast<std::chrono::milliseconds>(stop - start).count() << "[ms]" << std::endl;

        quicksort(globalMaxKHeap, 0, sizeofheap*NofCP-1);

        int count = 0;
        int g;
        for(g=0; g < sizeofheap*NofCP; g++){
            if(globalMaxKHeap[g] != DBL_MIN){
                count++;
            }
            if(count == NofCP){
                break;
            }
        }

        printf("The bound found by sample is: %.10f\n", globalMaxKHeap[g]);

        double bound = globalMaxKHeap[g];
        
        hipFree(d_globalMaxKHeap);
        hipFree(d_neccPairs);

        neccArray = findNeccPartitions(datasetP, datasetQ, indexesOfSplitsP, indexesOfSplitsQ, bound, neccArray, NofPartitions);
       
        printf("The total pair of partitions for calculations are: %d\n", neccArray.size());

        for(int i = 0; i < neccArray.size(); i++){
            h_neccArray[i] = neccArray[i];
        }

        //determine the number of blocks and number of threads depending on the 
        //neccessary pairs for calculations, that we extract based on bound
        int notUsefulThreadsId;
        int numberOfPairs = neccArray.size();
        if(numberOfPairs > THREADS_PER_BLOCK){
            threadsPerBlock = THREADS_PER_BLOCK;
            if(numberOfPairs % THREADS_PER_BLOCK != 0){
                blocksPerGrid = (numberOfPairs / THREADS_PER_BLOCK) + 1; 
                notUsefulThreadsId = numberOfPairs;
            }
            else{
                blocksPerGrid = (numberOfPairs / THREADS_PER_BLOCK);
                notUsefulThreadsId = -1;
            }
        }
        else{
            threadsPerBlock = numberOfPairs;
            blocksPerGrid = 1;
            notUsefulThreadsId = -1;
        }

        closestpairs* phase2_globalMaxKHeap = (closestpairs*)malloc(numberOfPairs*NofCP*sizeof(closestpairs));

        //start time of phase2
        auto startPhase2 = high_resolution_clock::now(); 

        hipMalloc(&d_neccArray, numberOfPairs*sizeof(pair));
        hipMalloc(&phase2_d_globalMaxKHeap, numberOfPairs*NofCP*sizeof(closestpairs));
        // Copy vectors from host memory to device memory
        hipMemcpy(d_neccArray, h_neccArray, numberOfPairs*sizeof(pair), hipMemcpyHostToDevice);
        
        //start time of phase2
        auto startPhase2Clean = high_resolution_clock::now(); 
        kernelPhase2<<<blocksPerGrid, threadsPerBlock>>>(d_datasetP, d_datasetQ, d_neccArray, NofCP, phase2_d_globalMaxKHeap, notUsefulThreadsId, bound, numberOfPairs);
        hipDeviceSynchronize();
        auto stopPhase2Clean = high_resolution_clock::now();

        std::cout << "CLEAN Time difference(Phase2 final) = " << std::chrono::duration_cast<std::chrono::milliseconds>(stopPhase2Clean - startPhase2Clean).count() << "[ms]" << std::endl;
        
        hipMemcpy(phase2_globalMaxKHeap, phase2_d_globalMaxKHeap, numberOfPairs*NofCP*sizeof(closestpairs), hipMemcpyDeviceToHost);
        
        auto stopPhase2 = high_resolution_clock::now();

        std::cout << "Time difference(Phase2 final) = " << std::chrono::duration_cast<std::chrono::milliseconds>(stopPhase2 - startPhase2).count() << "[ms]" << std::endl;
        //end time of phase2
        
        auto startSort = high_resolution_clock::now();

        quicksortPairs(phase2_globalMaxKHeap, 0, numberOfPairs*NofCP-1);
        //insertionSort(phase2_globalMaxKHeap, numberOfPairs*NofCP);
        //bubbleSort(phase2_globalMaxKHeap, numberOfPairs*NofCP);
        //selectionSort(phase2_globalMaxKHeap, numberOfPairs*NofCP);
        //mergeSort(phase2_globalMaxKHeap, 0, numberOfPairs*NofCP-1);

        auto stopSort = high_resolution_clock::now();

        std::cout << "Sort time = " << std::chrono::duration_cast<std::chrono::milliseconds>(stopSort - startSort).count() << "[ms]" << std::endl;

        //deleteSamePairs(phase2_globalMaxKHeap, numberOfPairs*NofCP);

        int i = 0;
        int counter = 0;
        while(counter < NofCP){
            if(phase2_globalMaxKHeap[i].dist != DBL_MIN){

                if(i != 0){
                    if(phase2_globalMaxKHeap[i].p.id != phase2_globalMaxKHeap[i-1].p.id || phase2_globalMaxKHeap[i].q.id != phase2_globalMaxKHeap[i-1].q.id){
                        counter++;
                        printf("The pair is: p (%f, %f) and q (%f, %f) with Distance: %.10lf\n", phase2_globalMaxKHeap[i].p.x, phase2_globalMaxKHeap[i].p.y, phase2_globalMaxKHeap[i].q.x, phase2_globalMaxKHeap[i].q.y, phase2_globalMaxKHeap[i].dist);
                    }    
                }
                else{
                    counter++;
                    printf("The pair is: p (%f, %f) and q (%f, %f) with Distance: %.10lf\n", phase2_globalMaxKHeap[i].p.x, phase2_globalMaxKHeap[i].p.y, phase2_globalMaxKHeap[i].q.x, phase2_globalMaxKHeap[i].q.y, phase2_globalMaxKHeap[i].dist);
                }
            
            }
            i++;            
        }

        // Free device memory
        hipFree(d_datasetQ);
        hipFree(d_datasetP);
        hipFree(d_neccArray);
        hipFree(phase2_d_globalMaxKHeap);
        
        auto stopTotal = high_resolution_clock::now();

        std::cout << "Total Execution Time = " << std::chrono::duration_cast<std::chrono::milliseconds>(stopTotal - startTotal).count() << "[ms]" << std::endl;
        
        
    }
    catch (std::exception & ex)
    {
        //report any exception
        std::cout << "Exception: " << ex.what() << std::endl;
        return 1;
    }


}
